
#include <hip/hip_runtime.h>
#include <iostream>
#include <sys/time.h>
#include <cstdlib>
#include <chrono>

/**
 * mergesort.cu
 * a one-file c++ / cuda program for performing mergesort on the GPU
 * While the program execution is fairly slow, most of its runnning time
 *  is spent allocating memory on the GPU.
 * For a more complex program that performs many calculations,
 *  running on the GPU may provide a significant boost in performance
 * 
 * This code was written by Kevin Albert and was obtained from his GitHub
 * at https://github.com/54kevinalbert/gpu-mergesort. I (Will Thompson)
 * modified the code for the assignment.
 */

// data[], size, threads, blocks, 
void mergesort(double*, long, dim3, dim3);
// A[]. B[], size, width, slices, nThreads
__global__ void gpu_mergesort(double*, double*, long, long, long, dim3*, dim3*);
__device__ void gpu_bottomUpMerge(double*, double*, long, long, long);

#define min(a, b) (a < b ? a : b)

int main(int argc, char** argv) {

    dim3 threadsPerBlock;
    dim3 blocksPerGrid;

    threadsPerBlock.x = 32;
    threadsPerBlock.y = 1;
    threadsPerBlock.z = 1;

    blocksPerGrid.x = 8;
    blocksPerGrid.y = 1;
    blocksPerGrid.z = 1;

    //
    // Read numbers from stdin
    //
	int size = atoi(argv[2]);;
    double* data = new double[size];
	srand(10);
	for(int i=0; i<size; ++i) data[i] = (double)(rand() % 100);

    // merge-sort the data
	auto start = std::chrono::steady_clock::now();
    mergesort(data, size, threadsPerBlock, blocksPerGrid);
	auto end = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - start);
	std::cout << "CUDA Mergesort Time: " << end.count() << " ms" << std::endl;

	bool isSorted = true;
	for (int i=0; i<size-1; ++i)
	{
		if(data[i] > data[i+1])
		{
			isSorted = false;
			break;
		}
	}
	std::cout << "The list is sorted: " << isSorted << std::endl;

}

void mergesort(double* data, long size, dim3 threadsPerBlock, dim3 blocksPerGrid) {

    //
    // Allocate two arrays on the GPU
    // we switch back and forth between them during the sort
    //
    double* D_data;
    double* D_swp;
    dim3* D_threads;
    dim3* D_blocks;
    
    // Actually allocate the two arrays
	hipMalloc((void**) &D_data, size * sizeof(double));
    hipMalloc((void**) &D_swp, size * sizeof(double));

    // Copy from our input list into the first array
    hipMemcpy(D_data, data, size * sizeof(double), hipMemcpyHostToDevice);
 
    //
    // Copy the thread / block info to the GPU as well
    //
	hipMalloc((void**) &D_threads, sizeof(dim3));
    hipMalloc((void**) &D_blocks, sizeof(dim3));

	hipMemcpy(D_threads, &threadsPerBlock, sizeof(dim3), hipMemcpyHostToDevice);
    hipMemcpy(D_blocks, &blocksPerGrid, sizeof(dim3), hipMemcpyHostToDevice);

    double* A = D_data;
    double* B = D_swp;

    long nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
                    blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;

    //
    // Slice up the list and give pieces of it to each thread, letting the pieces grow
    // bigger and bigger until the whole list is sorted
    //
    for (int width = 2; width < (size << 1); width <<= 1) {
        long slices = size / ((nThreads) * width) + 1;

        // Actually call the kernel
        gpu_mergesort<<<blocksPerGrid, threadsPerBlock>>>(A, B, size, width, slices, D_threads, D_blocks);

        // Switch the input / output arrays instead of copying them around
        A = A == D_data ? D_swp : D_data;
        B = B == D_data ? D_swp : D_data;
    }

    //
    // Get the list back from the GPU
    //
    hipMemcpy(data, A, size * sizeof(double), hipMemcpyDeviceToHost);
    
    // Free the GPU memory
	hipFree(A);
    hipFree(B);
}

// GPU helper function
// calculate the id of the current thread
__device__ unsigned int getIdx(dim3* threads, dim3* blocks) {
    int x;
    return threadIdx.x +
           threadIdx.y * (x  = threads->x) +
           threadIdx.z * (x *= threads->y) +
           blockIdx.x  * (x *= threads->z) +
           blockIdx.y  * (x *= blocks->z) +
           blockIdx.z  * (x *= blocks->y);
}

//
// Perform a full mergesort on our section of the data.
//
__global__ void gpu_mergesort(double* source, double* dest, long size, long width, long slices, dim3* threads, dim3* blocks) {
    unsigned int idx = getIdx(threads, blocks);
    long start = width*idx*slices, 
         middle, 
         end;

    for (long slice = 0; slice < slices; slice++) {
        if (start >= size)
            break;

        middle = min(start + (width >> 1), size);
        end = min(start + width, size);
        gpu_bottomUpMerge(source, dest, start, middle, end);
        start += width;
    }
}

//
// Finally, sort something
// gets called by gpu_mergesort() for each slice
//
__device__ void gpu_bottomUpMerge(double* source, double* dest, long start, long middle, long end) {
    long i = start;
    long j = middle;
    for (long k = start; k < end; k++) {
        if (i < middle && (j >= end || source[i] < source[j])) {
            dest[k] = source[i];
            i++;
        } else {
            dest[k] = source[j];
            j++;
        }
    }
}
