#include "hip/hip_runtime.h"
/**
 * -------------------- SOURCE -----------------------------------
 * Code: https://github.com/saigowri/CUDA/blob/master/quicksort.cu
 * Author: Sai Gowri
 * Date: July 15, 2016
 */

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <cstdlib>
#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>
#include <chrono>

int *r_values;
int *d_values;

float dataInitTime;
float correctnessTime;
float commSmallTime;
float commLargeTime;
float compSmallTime;
float compLargeTime;

const char *mainRegion = "main";
const char *parallel = "parallel";
const char *sequential = "sequential";
const char *genValuesTime = "data_init";
const char *barrier = "barrier";
const char *correctness = "correctness_check";
const char *comp = "comp";
const char *compSmall = "comp_small";
const char *compLarge = "comp_large";
const char *commRegion = "comm";
const char *commSmall = "comm_small";
const char *commLarge = "comm_large";

// Kernel function
__global__ static void quicksort(int *values, int N)
{
#define MAX_LEVELS 300

    int pivot, L, R;
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int start[MAX_LEVELS], end[MAX_LEVELS], i = 0;

    start[idx] = idx;
    end[idx] = N - 1;
    while (idx >= 0)
    {
        L = start[idx];
        R = end[idx];
        if (L < R)
        {
            pivot = values[L];
            while (L < R)
            {
                while (values[R] >= pivot && L < R)
                {
                    R--;
                }
                if (L < R)
                {
                    values[L++] = values[R];
                }
                while (values[L] <= pivot && L < R)
                {
                    L++;
                }
                if (L < R)
                {
                    values[R--] = values[L];
                }
            }
            values[L] = pivot;
            start[idx + 1] = L + 1;
            end[idx + 1] = end[idx];
            end[idx++] = L;

            if (end[idx] - start[idx] > end[idx - 1] - start[idx - 1])
            {
                int temp = start[idx];
                start[idx] = start[idx - 1];
                start[idx - 1] = temp;

                temp = end[idx];
                end[idx] = end[idx - 1];
                end[idx - 1] = temp;
            }
        }
        else
        {
            idx--;
        }
    }
}

int main(int argc, char **argv)
{
    CALI_CXX_MARK_FUNCTION;
    CALI_MARK_BEGIN(mainRegion);

    hipEvent_t dataInitStart, dataInitStop, correctnessStart, correctnessStop;
    hipEvent_t compSmallStart, compSmallStop, commSmallStart, commSmallStop;
    hipEvent_t compLargeStart, compLargeStop, commLargeStart, commLargeStop;
    size_t size = atoi(argv[1]); // CHANGE TO CLI ARG
    printf("./quicksort starting with %d numbers...\n", size * sizeof(int));
    const int MAX_THREADS = atoi(argv[2]); // CHANGE TO CLI ARG

    std::cout << "MAX_THREADS: " << MAX_THREADS << std::endl;

    // allocate host memory
    r_values = (int *)malloc(size * sizeof(int));

    // allocate device memory
    hipMalloc((void **)&d_values, size * sizeof(int));

    // allocate threads per block
    const unsigned int cThreadsPerBlock = 128; // CHANGE TO CLI ARG

    // Get dataset from command line
    // Generate random numbers
    hipEventCreate(&dataInitStart);
    hipEventCreate(&dataInitStop);
    hipEventCreate(&commSmallStart);
    hipEventCreate(&commSmallStop);
    hipEventCreate(&commLargeStart);
    hipEventCreate(&commLargeStop);
    hipEventCreate(&compSmallStart);
    hipEventCreate(&compSmallStop);
    hipEventCreate(&compLargeStart);
    hipEventCreate(&compLargeStop);
    hipEventCreate(&correctnessStart);
    hipEventCreate(&correctnessStop);

    CALI_MARK_BEGIN(genValuesTime);
    hipEventRecord(dataInitStart, 0);
    srand(time(NULL));
    for (int i = 0; i < size; i++)
    {
        r_values[i] = rand() % 100;
    }
    hipEventRecord(dataInitStop, 0);
    hipEventSynchronize(dataInitStop);
    CALI_MARK_END(genValuesTime);

    hipEventElapsedTime(&dataInitTime, dataInitStart, dataInitStop);

    // Copy data from host to device
    CALI_MARK_BEGIN(commRegion);
    CALI_MARK_BEGIN(commLarge);
    hipEventRecord(commLargeStart, 0);
    hipMemcpy(d_values, r_values, size * sizeof(int), hipMemcpyHostToDevice);
    hipEventRecord(commLargeStop, 0);
    hipEventSynchronize(commLargeStop);
    CALI_MARK_END(commLarge);
    CALI_MARK_END(commRegion);

    hipEventElapsedTime(&commLargeTime, commLargeStart, commLargeStop);

    // Start timer
    printf("Beginning kernel execution...\n");
    hipDeviceSynchronize();

    // Execute kernel
    CALI_MARK_BEGIN(comp);
    hipEventRecord(compLargeStart, 0);
    quicksort<<<MAX_THREADS / cThreadsPerBlock, MAX_THREADS / cThreadsPerBlock, cThreadsPerBlock>>>(d_values, size);
    hipEventRecord(compLargeStop, 0);
    hipEventSynchronize(compLargeStop);
    CALI_MARK_END(comp);

    hipEventElapsedTime(&compLargeTime, compLargeStart, compLargeStop);

    hipDeviceSynchronize();

    printf("\nKernel execution completed in %f ms\n", compLargeTime);

    // copy data back to host
    CALI_MARK_BEGIN(commRegion);
    CALI_MARK_BEGIN(commLarge);
    hipMemcpy(r_values, d_values, size * sizeof(int), hipMemcpyDeviceToHost);
    CALI_MARK_END(commLarge);
    CALI_MARK_END(commRegion);

    CALI_MARK_BEGIN("correctness");
    hipEventRecord(correctnessStart, 0);
    bool isSorted = true;
    for (int i = 0; i < size - 1; i++)
    {
        if (r_values[i] > r_values[i + 1])
        {
            isSorted = false;
            break;
        }
    }
    hipEventRecord(correctnessStop, 0);
    CALI_MARK_END("correctness");

    hipEventElapsedTime(&correctnessTime, correctnessStart, correctnessStop);

    if (isSorted)
    {
        printf("Array is sorted (LESSGO)\n");
    }
    else
    {
        printf("Array is not sorted (womp womp)\n");
    }

    // Print out all times
    printf("Data init time: %f ms\n", dataInitTime);
    printf("Comm large time: %f ms\n", commLargeTime);
    printf("Comp large time: %f ms\n", compLargeTime);
    printf("Correctness check time: %f ms\n", correctnessTime);
    // free memory
    hipEventDestroy(dataInitStart);
    hipEventDestroy(dataInitStop);
    hipEventDestroy(commSmallStart);
    hipEventDestroy(commSmallStop);
    hipEventDestroy(commLargeStart);
    hipEventDestroy(commLargeStop);
    hipEventDestroy(compSmallStart);
    hipEventDestroy(compSmallStop);
    hipEventDestroy(compLargeStart);
    hipEventDestroy(compLargeStop);
    hipEventDestroy(correctnessStart);
    hipEventDestroy(correctnessStop);
    free(r_values);
    hipFree(d_values);

    // exit
    hipDeviceReset();
    hipDeviceReset();

    CALI_MARK_END(mainRegion);
}