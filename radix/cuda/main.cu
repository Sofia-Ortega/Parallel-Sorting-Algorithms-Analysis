
// ADAPTED FROM: https://github.com/jackfly/radix-sort-cuda/blob/master/cuda_implementation/main.cu

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <algorithm>
#include <iostream>
#include <ctime>
#include <fstream>
#include <string> 
#include <sstream>
#include <math.h>
#include <time.h>

#include "radix_sort.h"

using namespace std;

const char* create_array = "create_array";
const char* radix_sort = "radix_sort";
const char* cudaMemcpy_host_to_device = "cudaMemcpy_host_to_device";
const char* cudaMemcpy_device_to_host = "cudaMemcpy_device_to_host";


void radixsort_gpu(unsigned int* h_in, unsigned int num, unsigned int num_threads, bool printArray = false)
{
    unsigned int* out_gpu = new unsigned int[num];
    
    unsigned int* d_in;
    unsigned int* d_out;
    hipMalloc(&d_in, sizeof(unsigned int) * num);
    hipMalloc(&d_out, sizeof(unsigned int) * num);

    CALI_MARK_BEGIN(cudaMemcpy_host_to_device);
    hipMemcpy(d_in, h_in, sizeof(unsigned int) * num, hipMemcpyHostToDevice);
    CALI_MARK_END(cudaMemcpy_host_to_device);

    CALI_MARK_BEGIN(radix_sort);
    radix_sort(d_out, d_in, num, num_threads);
    CALI_MARK_END(radix_sort);

    CALI_MARK_BEGIN(cudaMemcpy_device_to_host);
    hipMemcpy(out_gpu, d_out, sizeof(unsigned int) * num, hipMemcpyDeviceToHost);
    CALI_MARK_END(cudaMemcpy_device_to_host);

    if(printArray) {
      printf("------------- Sorted: ----------------\n");
      for(int i = 0; i < num; i++) {
        printf("%i\n", out_gpu[i]);
      }
      printf("--------------\n");
    }

    // check if sorted
    bool isSorted = true;
    for(int i = 1; i < num; i++) {
      if (out_gpu[i - 1] > out_gpu[i]) {
        isSorted = false;
        break;
      }
    }

    if(isSorted) {
      printf("[SUCCESS] Output Sorted\n");
    } else {
      printf("[FAILED] Output NOT Sorted\n");
    }

    hipFree(d_out);
    hipFree(d_in);

    delete[] out_gpu;
}

int main(int argc, char** argv)
{

    // argv:
    // 0          1            2                 3
    // radix_cuda num_threasds num_vals_to_sort  [optional: printArray]
    struct timespec start, stop;

    CALI_CXX_MARK_FUNCTION;


    // get user input
    if(argc != 3 && argc != 4) {
      printf("Incorrect argument usage\n");
      printf("radix_cuda num_threads num_vals_to_sort [optional: print_array]\n");
      return -1;
    }
    
    int num_threads = atoi(argv[1]);
    int n_values = atoi(argv[2]);
    bool printArray = false;

    if(argc == 4) {
      printArray = atoi(argv[3]);
    }

    printf("Sorting %i values with %i threads\n", n_values, num_threads);

    // create caliper ConfigManager object
    cali::ConfigManager mgr;
    mgr.start();

    // initialize local array
    unsigned int* numbers = new unsigned int[n_values];
    for(int i = 0; i < n_values; i++) {
      numbers[i] = (rand() % 10000) + 1;
    }


    // print array
    if(printArray) {
      for(int i = 0; i < n_values; i++) {
        printf("%i\n", numbers[i]);
      }
    }

    // sorting
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
    radixsort_gpu(numbers, n_values, num_threads, printArray);
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
    double dt = (stop.tv_sec - start.tv_sec) * 1e6 + (stop.tv_nsec - start.tv_nsec) / 1e3;    // in microseconds
    printf("@time of CUDA run:\t\t\t[%.3f] microseconds\n", dt);

    delete[] numbers;

    adiak::init(NULL);
    adiak::user();
    adiak::launchdate();
    adiak::libraries();
    adiak::cmdline();
    adiak::clustername();
    adiak::value("num_threads", num_threads);
    adiak::value("num_blocks", num_threads / n_values);
    adiak::value("num_vals", n_values);
    adiak::value("program_name", "cuda_radix_sort");
    adiak::value("datatype_size", sizeof(int));

    // Flush Caliper output
    mgr.stop();
    mgr.flush();


}